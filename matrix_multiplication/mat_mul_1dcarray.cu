
#include <hip/hip_runtime.h>
#include <iostream>
#define N 512

//
__global__ void mat_mul (double* res, double* mat1, double* mat2, int n)
{
    int bid_x = blockIdx.x;
    int tid_x = threadIdx.x;

    for (int row_index{0}; row_index < n; row_index++) {
        for (int col_index{0}; col_index < n; col_index++) {
            if (bid_x == row_index && tid_x == col_index) {
                int index = row_index*n + col_index;
                res[index] = 0;
                // compute dot product
                for (int k{0}; k < n; k++) {
                    int dot_index_1 = row_index*n + k;
                    int dot_index_2 = k*n + col_index;
                    res[index] += mat1[dot_index_1] * mat2[dot_index_2];
                }
            }
        }
    }    
}

int main()
{

    double* arr = (double*) malloc( (N*N) * sizeof(double) );
    double* brr = (double*) malloc( (N*N) * sizeof(double) );
    double* res = (double*) malloc( (N*N) * sizeof(double) );
    double* correct_res = (double*) malloc( (N*N) * sizeof(double) );
    
    // -- INITIALIZE --
    // Note that arr[i][j] is the same as *(*(arr+i)+j)
    for (int i{}; i<N; i++)
    {
        arr[i*N +(N-1-i)] = 1;
        brr[i*N +(N-1-i)] = 1;
        correct_res[i*N +(i)] = 1;
    }
    // -- END INITIALIZE --
        
    // -- TRANSFER MATRIX FROM HOST TO DEVICE --
    // Allocate device memory
    double* d_a;
    double* d_b;
    double* d_res;
    hipMalloc( (void**)&d_a, sizeof(double)*N*N );
    hipMalloc( (void**)&d_b, sizeof(double)*N*N );
    hipMalloc( (void**)&d_res, sizeof(double)*N*N );
    
    // Transfer array memory from host to device
    hipMemcpy(d_a, arr, sizeof(double) * N*N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, brr, sizeof(double) * N*N, hipMemcpyHostToDevice);
    
    // -- END TRANSFER MATRIX FROM HOST TO DEVICE --
    
    // Execute kernel
    mat_mul<<<N,N>>>(d_res, d_a, d_b, N);
    
    // Transfer array result from device to host
    hipMemcpy(res, d_res, sizeof(double) * N *N, hipMemcpyDeviceToHost);
    
    // -- BEGIN TEST -- 
    bool correct{true};
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            int index = i*N +j;
            if (res[index] != correct_res[index]) {
                correct = false;
                break;
            }
        }
    }
    if (correct)
    printf("example PASSED\n");
    else
        printf("example FAILED: wrong result\n"); 
    // -- END TEST -- 
        
    // Deallocate cuda memory
    hipFree( d_a );
    hipFree( d_b );
    hipFree( d_res );
    // Deallocate host memory so must free each double pointer array in each matrix
    free( arr );
    free( brr );
    free( res );
    free( correct_res );
    return 0;
}