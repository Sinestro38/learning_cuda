
#include <hip/hip_runtime.h>
#include <stdio.h>
#define N 10000000

// OLD SINGLE-THREADED VERSION
__global__ void vector_add(float *out, float *a, float *b, int n) {
    int index = 0;
    int stride = 1;
    for(int i = index; i < n; i += stride){
        out[i] = a[i] + b[i];
    }
}

int main(){
    float *a, *b, *out; 

    // Allocate memory
    a   = (float*) malloc(sizeof(float) * N);
    b   = (float*) malloc(sizeof(float) * N);
    out = (float*) malloc(sizeof(float) * N);

    // Initialize array
    for(int i = 0; i < N; i++){
        a[i] = 1.0f; b[i] = 2.0f;
    }

    // Print first 10 elements of out
    for (int i = 0; i < 10; i++){
        if (i == 9){
            printf("%.3f\n", out[i]);
            continue;
        }
        printf("%.3f, ", out[i]);
    }

    // Allocate device memory
    float* d_a;
    float* d_b;
    float* d_out;
    hipMalloc( (void**)&d_a, sizeof(float)*N );
    hipMalloc( (void**)&d_b, sizeof(float)*N );
    hipMalloc( (void**)&d_out, sizeof(float)*N );

    // Transfer array memory from host to device
    hipMemcpy(d_a, a, sizeof(float) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, sizeof(float) * N, hipMemcpyHostToDevice);
        
    // Main function
    vector_add<<<1, 512>>>(d_out, d_a, d_b, N);

    // Transfer array memory from device to host
    hipMemcpy(out, d_out, sizeof(float) * N, hipMemcpyDeviceToHost);
        
    // Print first 10 elements of out
     for (int i = 0; i < 10; i++){
        if (i == 9){
            printf("%.3f\n", out[i]);
            continue;
        }
        printf("%.3f, ", out[i]);
    }
 
    // Deallocate device memory
    hipFree( d_a );
    hipFree( d_b );
    hipFree( d_out );

    // Deallocate host memory
    free( a );
    free( b );
    free( out );
}